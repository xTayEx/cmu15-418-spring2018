#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <sstream>
#include <iostream>

#include "CycleTimer.h"

inline void fatalError(const std::string &err, const char *filename, int lineno) {
  std::stringstream pos, msg;
  pos << "In " << filename << ": " << lineno << "\n";
  msg << err << "\n";
  std::cerr << pos.str () << msg.str();
  hipDeviceReset();
  exit(1);
}

inline void checkCudaImpl(hipError_t status, const char *filename, int lineno) {
  std::stringstream err;
  if (status != hipSuccess) {
    err << "CUDA error: " << hipGetErrorString(status) << "\n";
    fatalError(err.str(), filename, lineno);
  }
}

#define checkCuda(status) checkCudaImpl(status, __FILE__, __LINE__)


extern float toBW(int bytes, float sec);

__global__ void saxpy_kernel(int N, float alpha, float *x, float *y,
                             float *result) {

  // compute overall index from position of thread in current block,
  // and given the block we are in
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < N)
    result[index] = alpha * x[index] + y[index];
}

void saxpyCuda(int N, float alpha, float *xarray, float *yarray,
               float *resultarray) {

  int totalBytes = sizeof(float) * 3 * N;

  // compute number of blocks and threads per block
  const int threadsPerBlock = 512;
  const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

  float *device_x;
  float *device_y;
  float *device_result;

  //
  // TODO allocate device memory buffers on the GPU using hipMalloc
  //
  checkCuda(hipMalloc(&device_x, sizeof(float) * N));
  checkCuda(hipMalloc(&device_y, sizeof(float) * N));
  checkCuda(hipMalloc(&device_result, sizeof(float) * N));

  // start timing after allocation of device memory
  double startTime = CycleTimer::currentSeconds();

  //
  // TODO copy input arrays to the GPU using hipMemcpy
  //
  hipMemcpy(device_x, xarray, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(device_y, yarray, sizeof(float) * N, hipMemcpyHostToDevice);

  double kernelStartTime = CycleTimer::currentSeconds();
  // run kernel
  saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y,
                                            device_result);
  hipDeviceSynchronize();
  double kernelEndTime = CycleTimer::currentSeconds();
  printf("Kernel time: %.3f ms\n", 1000.0f * (kernelEndTime - kernelStartTime));

  //
  // TODO copy result from GPU using hipMemcpy
  //
  hipMemcpy(resultarray, device_result, sizeof(float) * N, hipMemcpyDeviceToHost);

  // end timing after result has been copied back into host memory
  double endTime = CycleTimer::currentSeconds();

  hipError_t errCode = hipPeekAtLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode,
            hipGetErrorString(errCode));
  }

  double overallDuration = endTime - startTime;
  printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration,
         toBW(totalBytes, overallDuration));

  // TODO free memory buffers on the GPU
  hipFree(device_x);
  hipFree(device_y);
  hipFree(device_result);
}

void printCudaInfo() {

  // for fun, just print out some stats on the machine

  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");
}
